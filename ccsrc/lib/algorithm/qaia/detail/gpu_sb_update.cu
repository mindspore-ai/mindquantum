#include "hip/hip_runtime.h"
/**
 * Copyright (c) Huawei Technologies Co., Ltd. 2022. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hipblas.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>

#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>

#include "algorithm/qaia/csr_base.h"
#include "algorithm/qaia/detail/common.cuh"
#include "algorithm/qaia/detail/gpu_sb.cuh"
#include "algorithm/qaia/detail/para.h"
#include "algorithm/qaia/detail/tools.cuh"

using mindquantum::Index;

namespace mindquantum::algorithm::qaia::detail {

void SBBase::dSB_update_int8(mindquantum::sparse::CsrBase<double> csr, double* x, Para paras) {
    Index* indptr = csr.indptr_;
    Index* indices = csr.indices_;
    double* data = csr.data_;
    int N = csr.dim_;
    int B = paras.B;
    float xi = paras.xi;
    float delta = paras.delta;
    float dt = paras.dt;
    int n_iter = paras.n_iter;
    int NN = N * N;
    int NB = N * B;

    std::vector<int8_t> h_J(NN, 0);
    std::vector<int8_t> h_x(NB);

    fill_J<int8_t>(indices, indptr, data, &h_J, N);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < B; j++) {
            h_x[j * N + i] = static_cast<int8_t>(round(x[i * B + j] * 127.0));
        }
    }

    int8_t *d_J, *d_x, *signx;
    int *d_y, *tmp;
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_J), NN * sizeof(int8_t)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_x), NB * sizeof(int8_t)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_y), NB * sizeof(int)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&tmp), NB * sizeof(int)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&signx), NB * sizeof(int8_t)));
    HANDLE_ERROR(hipMemcpy(d_J, h_J.data(), NN * sizeof(int8_t), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_x, h_x.data(), NB * sizeof(int8_t), hipMemcpyHostToDevice));

    init_xy<int><<<(NB + 255) / 256, 256>>>(d_y, NB, time(NULL));

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    int alpha = 1;
    int b = 0;
    for (int i = 0; i < n_iter; i++) {
        float beta = (n_iter == 1) ? -delta : static_cast<float>(i) / (n_iter - 1) - delta;

        sign_kernel<int8_t><<<(NB + 255) / 256, 256>>>(d_x, signx, NB);

        CHECK_CUBLAS(hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, B, N, &alpha, d_J, HIP_R_8I, N, signx,
                                  HIP_R_8I, N, &b, tmp, HIP_R_32I, N, HIP_R_32I, CUBLAS_GEMM_ALGO15_TENSOR_OP));

        update_tail<<<(NB + 255) / 256, 256>>>(tmp, d_x, d_y, NB, xi, beta, delta, dt);
    }
    CHECK_CUDA(hipMemcpy(h_x.data(), d_x, NB * sizeof(int8_t), hipMemcpyDeviceToHost))

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < B; j++) {
            x[i * B + j] = static_cast<double>(h_x[j * N + i]) / 127.0;
        }
    }

    HANDLE_ERROR(hipFree(d_J));
    HANDLE_ERROR(hipFree(d_x));
    HANDLE_ERROR(hipFree(d_y));
    HANDLE_ERROR(hipFree(tmp));
    HANDLE_ERROR(hipFree(signx));
}

void SBBase::bSB_update_int8(mindquantum::sparse::CsrBase<double> csr, double* x, Para paras) {
    Index* indptr = csr.indptr_;
    Index* indices = csr.indices_;
    double* data = csr.data_;
    int N = csr.dim_;
    int B = paras.B;
    float xi = paras.xi;
    float delta = paras.delta;
    float dt = paras.dt;
    int n_iter = paras.n_iter;
    int NN = N * N;
    int NB = N * B;

    std::vector<int8_t> h_J(NN, 0);
    std::vector<int8_t> h_x(NB);

    fill_J<int8_t>(indices, indptr, data, &h_J, N);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < B; j++) {
            h_x[j * N + i] = static_cast<int8_t>(round(x[i * B + j] * 127.0));
        }
    }

    int8_t *d_J, *d_x;
    int *d_y, *tmp;
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_J), NN * sizeof(int8_t)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_x), NB * sizeof(int8_t)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_y), NB * sizeof(int)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&tmp), NB * sizeof(int)));
    HANDLE_ERROR(hipMemcpy(d_J, h_J.data(), NN * sizeof(int8_t), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_x, h_x.data(), NB * sizeof(int8_t), hipMemcpyHostToDevice));

    init_xy<int><<<(NB + 255) / 256, 256>>>(d_y, NB, time(NULL));

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    int alpha = 1;
    int b = 0;
    for (int i = 0; i < n_iter; i++) {
        float beta = (n_iter == 1) ? -delta : static_cast<float>(i) / (n_iter - 1) - delta;

        CHECK_CUBLAS(hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, B, N, &alpha, d_J, HIP_R_8I, N, d_x, HIP_R_8I,
                                  N, &b, tmp, HIP_R_32I, N, HIP_R_32I, CUBLAS_GEMM_ALGO15_TENSOR_OP));

        update_tail<<<(NB + 255) / 256, 256>>>(tmp, d_x, d_y, NB, xi, beta, delta, dt);
    }
    CHECK_CUDA(hipMemcpy(h_x.data(), d_x, NB * sizeof(int8_t), hipMemcpyDeviceToHost))

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < B; j++) {
            x[i * B + j] = static_cast<double>(h_x[j * N + i]) / 127.0;
        }
    }
    HANDLE_ERROR(hipFree(d_J));
    HANDLE_ERROR(hipFree(d_x));
    HANDLE_ERROR(hipFree(d_y));
    HANDLE_ERROR(hipFree(tmp));
}

void SBBase::bSB_update_fp16(mindquantum::sparse::CsrBase<double> csr, double* x, Para paras) {
    Index* indptr = csr.indptr_;
    Index* indices = csr.indices_;
    double* data = csr.data_;
    int N = csr.dim_;
    int B = paras.B;
    float xi = paras.xi;
    float delta = paras.delta;
    float dt = paras.dt;
    int n_iter = paras.n_iter;
    int NN = N * N;
    int NB = N * B;

    std::vector<half> h_J(NN, 0);
    std::vector<half> h_x(NB);

    fill_J<half>(indices, indptr, data, &h_J, N);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < B; j++) {
            h_x[i * B + j] = static_cast<half>(x[i * B + j]);
        }
    }

    half *d_J, *d_x;
    half *d_y, *tmp;
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_J), NN * sizeof(half)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_x), NB * sizeof(half)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_y), NB * sizeof(half)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&tmp), NB * sizeof(half)));
    HANDLE_ERROR(hipMemcpy(d_J, h_J.data(), NN * sizeof(half), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_x, h_x.data(), NB * sizeof(half), hipMemcpyHostToDevice));

    init_xy<half><<<(NB + 255) / 256, 256>>>(d_y, NB, time(NULL));

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    // hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
    half b = 0.0;
    half alpha = 1.0;
    for (int i = 0; i < n_iter; i++) {
        float beta = (n_iter == 1) ? -delta : static_cast<float>(i) / (n_iter - 1) - delta;

        CHECK_CUBLAS(hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, B, N, N, &alpha, d_x, HIP_R_16F, B, d_J,
                                  HIP_R_16F, N, &b, tmp, HIP_R_16F, B, HIPBLAS_COMPUTE_16F,
                                  CUBLAS_GEMM_ALGO15_TENSOR_OP));

        update_tail_half<<<(NB + 255) / 256, 256>>>(tmp, d_x, d_y, NB, xi, beta, delta, dt);
    }
    CHECK_CUDA(hipMemcpy(h_x.data(), d_x, NB * sizeof(half), hipMemcpyDeviceToHost))

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < B; j++) {
            x[i * B + j] = static_cast<double>(h_x[i * B + j]);
        }
    }

    HANDLE_ERROR(hipFree(d_J));
    HANDLE_ERROR(hipFree(d_x));
    HANDLE_ERROR(hipFree(d_y));
    HANDLE_ERROR(hipFree(tmp));
}

void SBBase::dSB_update_fp16(mindquantum::sparse::CsrBase<double> csr, double* x, Para paras) {
    Index* indptr = csr.indptr_;
    Index* indices = csr.indices_;
    double* data = csr.data_;
    int N = csr.dim_;
    int B = paras.B;
    float xi = paras.xi;
    float delta = paras.delta;
    float dt = paras.dt;
    int n_iter = paras.n_iter;
    int NN = N * N;
    int NB = N * B;

    std::vector<half> h_J(NN, 0);
    std::vector<half> h_x(NB);

    fill_J<half>(indices, indptr, data, &h_J, N);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < B; j++) {
            h_x[i * B + j] = static_cast<half>(x[i * B + j]);
        }
    }

    half *d_J, *d_x;
    half *d_y, *tmp, *signx;
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_J), NN * sizeof(half)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_x), NB * sizeof(half)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_y), NB * sizeof(half)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&tmp), NB * sizeof(half)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&signx), NB * sizeof(half)));
    HANDLE_ERROR(hipMemcpy(d_J, h_J.data(), NN * sizeof(half), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_x, h_x.data(), NB * sizeof(half), hipMemcpyHostToDevice));

    init_xy<half><<<(NB + 255) / 256, 256>>>(d_y, NB, time(NULL));

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    half b = 0.0;
    half alpha = 1.0;
    for (int i = 0; i < n_iter; i++) {
        float beta = (n_iter == 1) ? -delta : static_cast<float>(i) / (n_iter - 1) - delta;

        sign_kernel<half><<<(NB + 255) / 256, 256>>>(d_x, signx, NB);

        CHECK_CUBLAS(hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, B, N, N, &alpha, signx, HIP_R_16F, B, d_J,
                                  HIP_R_16F, N, &b, tmp, HIP_R_16F, B, HIPBLAS_COMPUTE_16F,
                                  CUBLAS_GEMM_ALGO15_TENSOR_OP));

        update_tail_half<<<(NB + 255) / 256, 256>>>(tmp, d_x, d_y, NB, xi, beta, delta, dt);
    }
    CHECK_CUDA(hipMemcpy(h_x.data(), d_x, NB * sizeof(half), hipMemcpyDeviceToHost))

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < B; j++) {
            x[i * B + j] = static_cast<double>(h_x[i * B + j]);
        }
    }
    HANDLE_ERROR(hipFree(d_J));
    HANDLE_ERROR(hipFree(d_x));
    HANDLE_ERROR(hipFree(d_y));
    HANDLE_ERROR(hipFree(tmp));
    HANDLE_ERROR(hipFree(signx));
}

void SBBase::dSB_update_h_int8(mindquantum::sparse::CsrBase<double> csr, double* x, Para paras, double* h, int h_size) {
    Index* indptr = csr.indptr_;
    Index* indices = csr.indices_;
    double* data = csr.data_;
    int N = csr.dim_;
    int B = paras.B;
    float xi = paras.xi;
    float delta = paras.delta;
    float dt = paras.dt;
    int n_iter = paras.n_iter;
    int NN = N * N;
    int NB = N * B;

    if (h_size != N * B) {
        throw std::runtime_error("External field h must have length N * batch_size(" + std::to_string(N * B)
                                 + "), but got length " + std::to_string(h_size));
    }

    std::vector<int8_t> h_J(NN, 0);
    std::vector<int8_t> h_x(NB);

    fill_J<int8_t>(indices, indptr, data, &h_J, N);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < B; j++) {
            h_x[j * N + i] = static_cast<int8_t>(round(x[i * B + j] * 127.0));
        }
    }

    int8_t *d_J, *d_x, *signx;
    int *d_y, *tmp, *d_h;
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_J), NN * sizeof(int8_t)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_x), NB * sizeof(int8_t)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_y), NB * sizeof(int)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&tmp), NB * sizeof(int)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&signx), NB * sizeof(int8_t)));
    HANDLE_ERROR(hipMemcpy(d_J, h_J.data(), NN * sizeof(int8_t), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_x, h_x.data(), NB * sizeof(int8_t), hipMemcpyHostToDevice));

    std::vector<int> h_h(NB, 0);
    for (int i = 0; i < NB; i++) {
        h_h[i] = static_cast<int>(round(h[i] * 127.0));
    }

    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_h), NB * sizeof(int)));
    HANDLE_ERROR(hipMemcpy(d_h, h_h.data(), NB * sizeof(int), hipMemcpyHostToDevice));

    init_xy<int><<<(NB + 255) / 256, 256>>>(d_y, NB, time(NULL));

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    int alpha = 1;
    int b = 0;
    for (int i = 0; i < n_iter; i++) {
        float beta = (n_iter == 1) ? -delta : static_cast<float>(i) / (n_iter - 1) - delta;

        sign_kernel<int8_t><<<(NB + 255) / 256, 256>>>(d_x, signx, NB);

        CHECK_CUBLAS(hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, B, N, &alpha, d_J, HIP_R_8I, N, signx,
                                  HIP_R_8I, N, &b, tmp, HIP_R_32I, N, HIP_R_32I, CUBLAS_GEMM_ALGO15_TENSOR_OP));

        update_h_tail<<<(NB + 255) / 256, 256>>>(tmp, d_x, d_y, d_h, NB, xi, beta, delta, dt);
    }
    CHECK_CUDA(hipMemcpy(h_x.data(), d_x, NB * sizeof(int8_t), hipMemcpyDeviceToHost))

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < B; j++) {
            x[i * B + j] = static_cast<double>(h_x[j * N + i]) / 127.0;
        }
    }
    HANDLE_ERROR(hipFree(d_h));
    HANDLE_ERROR(hipFree(d_J));
    HANDLE_ERROR(hipFree(d_x));
    HANDLE_ERROR(hipFree(d_y));
    HANDLE_ERROR(hipFree(tmp));
    HANDLE_ERROR(hipFree(signx));
}

void SBBase::bSB_update_h_int8(mindquantum::sparse::CsrBase<double> csr, double* x, Para paras, double* h, int h_size) {
    Index* indptr = csr.indptr_;
    Index* indices = csr.indices_;
    double* data = csr.data_;
    int N = csr.dim_;
    int B = paras.B;
    float xi = paras.xi;
    float delta = paras.delta;
    float dt = paras.dt;
    int n_iter = paras.n_iter;
    int NN = N * N;
    int NB = N * B;

    if (h_size != N * B) {
        throw std::runtime_error("External field h must have length N * batch_size(" + std::to_string(N * B)
                                 + "), but got length " + std::to_string(h_size));
    }

    std::vector<int8_t> h_J(NN, 0);
    std::vector<int8_t> h_x(NB);

    fill_J<int8_t>(indices, indptr, data, &h_J, N);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < B; j++) {
            h_x[j * N + i] = static_cast<int8_t>(round(x[i * B + j] * 127.0));
        }
    }

    int8_t *d_J, *d_x;
    int *d_y, *tmp, *d_h;
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_J), NN * sizeof(int8_t)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_x), NB * sizeof(int8_t)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_y), NB * sizeof(int)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&tmp), NB * sizeof(int)));
    HANDLE_ERROR(hipMemcpy(d_J, h_J.data(), NN * sizeof(int8_t), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_x, h_x.data(), NB * sizeof(int8_t), hipMemcpyHostToDevice));

    std::vector<int> h_h(NB, 0);
    for (int i = 0; i < NB; i++) {
        h_h[i] = static_cast<int>(round(h[i] * 127.0));
    }

    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_h), NB * sizeof(int)));
    HANDLE_ERROR(hipMemcpy(d_h, h_h.data(), NB * sizeof(int), hipMemcpyHostToDevice));

    init_xy<int><<<(NB + 255) / 256, 256>>>(d_y, NB, time(NULL));

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    int alpha = 1;
    int b = 0;
    for (int i = 0; i < n_iter; i++) {
        float beta = (n_iter == 1) ? -delta : static_cast<float>(i) / (n_iter - 1) - delta;

        CHECK_CUBLAS(hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, B, N, &alpha, d_J, HIP_R_8I, N, d_x, HIP_R_8I,
                                  N, &b, tmp, HIP_R_32I, N, HIP_R_32I, CUBLAS_GEMM_ALGO15_TENSOR_OP));

        update_h_tail<<<(NB + 255) / 256, 256>>>(tmp, d_x, d_y, d_h, NB, xi, beta, delta, dt);
    }
    CHECK_CUDA(hipMemcpy(h_x.data(), d_x, NB * sizeof(int8_t), hipMemcpyDeviceToHost))

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < B; j++) {
            x[i * B + j] = static_cast<double>(h_x[j * N + i]) / 127.0;
        }
    }

    HANDLE_ERROR(hipFree(d_h));
    HANDLE_ERROR(hipFree(d_J));
    HANDLE_ERROR(hipFree(d_x));
    HANDLE_ERROR(hipFree(d_y));
    HANDLE_ERROR(hipFree(tmp));
}

void SBBase::bSB_update_h_fp16(mindquantum::sparse::CsrBase<double> csr, double* x, Para paras, double* h, int h_size) {
    Index* indptr = csr.indptr_;
    Index* indices = csr.indices_;
    double* data = csr.data_;
    int N = csr.dim_;
    int B = paras.B;
    float xi = paras.xi;
    float delta = paras.delta;
    float dt = paras.dt;
    int n_iter = paras.n_iter;
    int NN = N * N;
    int NB = N * B;

    if (h_size != N * B) {
        throw std::runtime_error("External field h must have length N * batch_size(" + std::to_string(N * B)
                                 + "), but got length " + std::to_string(h_size));
    }

    std::vector<half> h_J(NN, 0);
    std::vector<half> h_x(NB);

    fill_J<half>(indices, indptr, data, &h_J, N);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < B; j++) {
            h_x[i * B + j] = static_cast<half>(x[i * B + j]);
        }
    }

    half *d_J, *d_x;
    half *d_y, *tmp, *d_h;
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_J), NN * sizeof(half)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_x), NB * sizeof(half)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_y), NB * sizeof(half)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&tmp), NB * sizeof(half)));
    HANDLE_ERROR(hipMemcpy(d_J, h_J.data(), NN * sizeof(half), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_x, h_x.data(), NB * sizeof(half), hipMemcpyHostToDevice));

    std::vector<half> h_h(NB, 0);
    for (int i = 0; i < NB; i++) {
        h_h[i] = static_cast<half>(h[i]);
    }

    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_h), NB * sizeof(half)));
    HANDLE_ERROR(hipMemcpy(d_h, h_h.data(), NB * sizeof(half), hipMemcpyHostToDevice));

    init_xy<half><<<(NB + 255) / 256, 256>>>(d_y, NB, time(NULL));

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    // hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
    half b = 0.0;
    half alpha = 1.0;
    for (int i = 0; i < n_iter; i++) {
        float beta = (n_iter == 1) ? -delta : static_cast<float>(i) / (n_iter - 1) - delta;

        CHECK_CUBLAS(hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, B, N, N, &alpha, d_x, HIP_R_16F, B, d_J,
                                  HIP_R_16F, N, &b, tmp, HIP_R_16F, B, HIPBLAS_COMPUTE_16F,
                                  CUBLAS_GEMM_ALGO15_TENSOR_OP));

        update_h_tail_half<<<(NB + 255) / 256, 256>>>(tmp, d_x, d_y, d_h, NB, xi, beta, delta, dt);
    }
    CHECK_CUDA(hipMemcpy(h_x.data(), d_x, NB * sizeof(half), hipMemcpyDeviceToHost))

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < B; j++) {
            x[i * B + j] = static_cast<double>(h_x[i * B + j]);
        }
    }

    HANDLE_ERROR(hipFree(d_h));
    HANDLE_ERROR(hipFree(d_J));
    HANDLE_ERROR(hipFree(d_x));
    HANDLE_ERROR(hipFree(d_y));
    HANDLE_ERROR(hipFree(tmp));
}

void SBBase::dSB_update_h_fp16(mindquantum::sparse::CsrBase<double> csr, double* x, Para paras, double* h, int h_size) {
    Index* indptr = csr.indptr_;
    Index* indices = csr.indices_;
    double* data = csr.data_;
    int N = csr.dim_;
    int B = paras.B;
    float xi = paras.xi;
    float delta = paras.delta;
    float dt = paras.dt;
    int n_iter = paras.n_iter;
    int NN = N * N;
    int NB = N * B;

    if (h_size != N * B) {
        throw std::runtime_error("External field h must have length N * batch_size(" + std::to_string(N * B)
                                 + "), but got length " + std::to_string(h_size));
    }

    std::vector<half> h_J(NN, 0);
    std::vector<half> h_x(NB);

    fill_J<half>(indices, indptr, data, &h_J, N);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < B; j++) {
            h_x[i * B + j] = static_cast<half>(x[i * B + j]);
        }
    }

    half *d_J, *d_x;
    half *d_y, *tmp, *d_h, *signx;
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_J), NN * sizeof(half)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_x), NB * sizeof(half)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_y), NB * sizeof(half)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&tmp), NB * sizeof(half)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&signx), NB * sizeof(half)));
    HANDLE_ERROR(hipMemcpy(d_J, h_J.data(), NN * sizeof(half), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_x, h_x.data(), NB * sizeof(half), hipMemcpyHostToDevice));

    std::vector<half> h_h(NB, 0);
    for (int i = 0; i < NB; i++) {
        h_h[i] = static_cast<half>(h[i]);
    }

    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_h), NB * sizeof(half)));
    HANDLE_ERROR(hipMemcpy(d_h, h_h.data(), NB * sizeof(half), hipMemcpyHostToDevice));

    init_xy<half><<<(NB + 255) / 256, 256>>>(d_y, NB, time(NULL));

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    half b = 0.0;
    half alpha = 1.0;
    for (int i = 0; i < n_iter; i++) {
        float beta = (n_iter == 1) ? -delta : static_cast<float>(i) / (n_iter - 1) - delta;

        sign_kernel<half><<<(NB + 255) / 256, 256>>>(d_x, signx, NB);

        CHECK_CUBLAS(hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, B, N, N, &alpha, signx, HIP_R_16F, B, d_J,
                                  HIP_R_16F, N, &b, tmp, HIP_R_16F, B, HIPBLAS_COMPUTE_16F,
                                  CUBLAS_GEMM_ALGO15_TENSOR_OP));

        update_h_tail_half<<<(NB + 255) / 256, 256>>>(tmp, d_x, d_y, d_h, NB, xi, beta, delta, dt);
    }
    CHECK_CUDA(hipMemcpy(h_x.data(), d_x, NB * sizeof(half), hipMemcpyDeviceToHost))

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < B; j++) {
            x[i * B + j] = static_cast<double>(h_x[i * B + j]);
        }
    }

    HANDLE_ERROR(hipFree(d_h));
    HANDLE_ERROR(hipFree(d_J));
    HANDLE_ERROR(hipFree(d_x));
    HANDLE_ERROR(hipFree(d_y));
    HANDLE_ERROR(hipFree(tmp));
    HANDLE_ERROR(hipFree(signx));
}

void SBBase::cublas_warmup(int N, int B) {
    int8_t *d_J, *d_x;
    int* tmp;
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_J), N * N * sizeof(int8_t)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_x), N * B * sizeof(int8_t)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&tmp), N * B * sizeof(int)));
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    int alpha = 1;
    int beta = 0;
    for (int i = 0; i < 100; i++) {
        CHECK_CUBLAS(hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, B, N, &alpha, d_J, HIP_R_8I, N, d_x, HIP_R_8I,
                                  N, &beta, tmp, HIP_R_32I, N, HIP_R_32I, CUBLAS_GEMM_ALGO15_TENSOR_OP));
    }
    HANDLE_ERROR(hipFree(d_J));
    HANDLE_ERROR(hipFree(d_x));
    HANDLE_ERROR(hipFree(tmp));
}

}  // namespace mindquantum::algorithm::qaia::detail
