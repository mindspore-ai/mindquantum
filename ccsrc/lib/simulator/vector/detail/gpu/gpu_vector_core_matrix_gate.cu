/**
 * Copyright (c) Huawei Technologies Co., Ltd. 2022. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <thrust/transform_reduce.h>

#include "config/openmp.h"
#include "simulator/utils.h"
#include "simulator/vector/detail/cuquantum_vector_double_policy.cuh"
#include "simulator/vector/detail/cuquantum_vector_float_policy.cuh"
#include "simulator/vector/detail/gpu_vector_double_policy.cuh"
#include "simulator/vector/detail/gpu_vector_float_policy.cuh"
#include "simulator/vector/detail/gpu_vector_policy.cuh"
#include "thrust/device_ptr.h"
#include "thrust/device_vector.h"
#include "thrust/functional.h"
#include "thrust/inner_product.h"
namespace mindquantum::sim::vector::detail {
template <typename derived_, typename calc_type_>
void GPUVectorPolicyBase<derived_, calc_type_>::ApplyNQubitsMatrix(const qs_data_p_t& src_out, qs_data_p_t* des_p,
                                                                   const qbits_t& objs, const qbits_t& ctrls,
                                                                   const std::vector<std::vector<py_qs_data_t>>& gate,
                                                                   index_t dim) {
    auto& des = *des_p;
    if (des == nullptr) {
        des = derived::InitState(dim);
    }
    qs_data_p_t src;
    bool will_free = false;
    if (src_out == nullptr) {
        src = derived::InitState(dim);
        will_free = true;
    } else {
        src = src_out;
    }
    size_t n_qubit = objs.size();
    size_t m_dim = (static_cast<uint64_t>(1) << n_qubit);
    size_t ctrl_mask = 0;
    for (auto& i : ctrls) {
        ctrl_mask |= static_cast<uint64_t>(1) << i;
    }
    std::vector<size_t> obj_masks{};
    for (size_t i = 0; i < m_dim; i++) {
        size_t n = 0;
        size_t mask_j = 0;
        for (size_t j = i; j != 0; j >>= 1) {
            if (j & 1) {
                mask_j += static_cast<uint64_t>(1) << objs[n];
            }
            n += 1;
        }
        obj_masks.push_back(mask_j);
    }
    auto obj_mask = obj_masks.back();
    thrust::device_vector<size_t> device_obj_masks = obj_masks;
    auto device_obj_masks_ptr = thrust::raw_pointer_cast(device_obj_masks.data());
    thrust::device_vector<qs_data_t> device_gate;
    for (auto& m : gate) {
        for (auto v : m) {
            device_gate.push_back(v);
        }
    }
    auto device_gate_ptr = thrust::raw_pointer_cast(device_gate.data());

    thrust::counting_iterator<size_t> l(0);
    if (des == src_out) {
        thrust::for_each(l, l + dim, [=] __device__(size_t l) {
            if (((l & ctrl_mask) == ctrl_mask) && ((l & obj_mask) == 0)) {
                qs_data_p_t res_tmp = (qs_data_p_t) malloc(sizeof(qs_data_t) * m_dim);
                for (size_t i = 0; i < m_dim; i++) {
                    qs_data_t tmp = 0;
                    for (size_t j = 0; j < m_dim; j++) {
                        tmp += device_gate_ptr[i * m_dim + j] * src[device_obj_masks_ptr[j] | l];
                    }
                    res_tmp[i] = tmp;
                }
                for (size_t i = 0; i < m_dim; i++) {
                    des[device_obj_masks_ptr[i] | l] = res_tmp[i];
                }
            }
        });
    } else {
        thrust::for_each(l, l + dim, [=] __device__(size_t l) {
            if (((l & ctrl_mask) == ctrl_mask) && ((l & obj_mask) == 0)) {
                for (size_t i = 0; i < m_dim; i++) {
                    qs_data_t tmp = 0;
                    for (size_t j = 0; j < m_dim; j++) {
                        tmp += device_gate_ptr[i * m_dim + j] * src[device_obj_masks_ptr[j] | l];
                    }
                    des[device_obj_masks_ptr[i] | l] = tmp;
                }
            }
        });
    }
    if (will_free) {
        derived::FreeState(&src);
    }
}

template <typename derived_, typename calc_type_>
void GPUVectorPolicyBase<derived_, calc_type_>::ApplyTwoQubitsMatrix(const qs_data_p_t& src_out, qs_data_p_t* des_p,
                                                                     const qbits_t& objs, const qbits_t& ctrls,
                                                                     const std::vector<std::vector<py_qs_data_t>>& m,
                                                                     index_t dim) {
    auto& des = *des_p;
    if (des == nullptr) {
        des = derived::InitState(dim);
    }
    qs_data_p_t src;
    bool will_free = false;
    if (src_out == nullptr) {
        src = derived::InitState(dim);
        will_free = true;
    } else {
        src = src_out;
    }
    DoubleQubitGateMask mask(objs, ctrls);
    qs_data_t m00 = m[0][0];
    qs_data_t m01 = m[0][1];
    qs_data_t m02 = m[0][2];
    qs_data_t m03 = m[0][3];
    qs_data_t m10 = m[1][0];
    qs_data_t m11 = m[1][1];
    qs_data_t m12 = m[1][2];
    qs_data_t m13 = m[1][3];
    qs_data_t m20 = m[2][0];
    qs_data_t m21 = m[2][1];
    qs_data_t m22 = m[2][2];
    qs_data_t m23 = m[2][3];
    qs_data_t m30 = m[3][0];
    qs_data_t m31 = m[3][1];
    qs_data_t m32 = m[3][2];
    qs_data_t m33 = m[3][3];
    auto obj_high_mask = mask.obj_high_mask;
    auto obj_rev_high_mask = mask.obj_rev_high_mask;
    auto obj_low_mask = mask.obj_low_mask;
    auto obj_rev_low_mask = mask.obj_rev_low_mask;
    size_t mask1 = (static_cast<uint64_t>(1) << objs[0]);
    size_t mask2 = (static_cast<uint64_t>(1) << objs[1]);
    auto ctrl_mask = mask.ctrl_mask;
    auto obj_mask = mask.obj_mask;
    thrust::counting_iterator<size_t> l(0);
    if (!mask.ctrl_mask) {
        thrust::for_each(l, l + (dim / 4), [=] __device__(size_t l) {
            index_t i;
            SHIFT_BIT_TWO(obj_low_mask, obj_rev_low_mask, obj_high_mask, obj_rev_high_mask, l, i);
            auto j = i + mask1;
            auto k = i + mask2;
            auto m = i + obj_mask;
            auto v00 = m00 * src[i] + m01 * src[j] + m02 * src[k] + m03 * src[m];
            auto v01 = m10 * src[i] + m11 * src[j] + m12 * src[k] + m13 * src[m];
            auto v10 = m20 * src[i] + m21 * src[j] + m22 * src[k] + m23 * src[m];
            auto v11 = m30 * src[i] + m31 * src[j] + m32 * src[k] + m33 * src[m];
            src[i] = v00;
            src[j] = v01;
            src[k] = v10;
            src[m] = v11;
        });
    } else {
        thrust::for_each(l, l + (dim / 4), [=] __device__(size_t l) {
            index_t i;
            SHIFT_BIT_TWO(obj_low_mask, obj_rev_low_mask, obj_high_mask, obj_rev_high_mask, l, i);
            if ((i & ctrl_mask) == ctrl_mask) {
                auto j = i + mask1;
                auto k = i + mask2;
                auto m = i + obj_mask;
                auto v00 = m00 * src[i] + m01 * src[j] + m02 * src[k] + m03 * src[m];
                auto v01 = m10 * src[i] + m11 * src[j] + m12 * src[k] + m13 * src[m];
                auto v10 = m20 * src[i] + m21 * src[j] + m22 * src[k] + m23 * src[m];
                auto v11 = m30 * src[i] + m31 * src[j] + m32 * src[k] + m33 * src[m];
                src[i] = v00;
                src[j] = v01;
                src[k] = v10;
                src[m] = v11;
            }
        });
    }
    if (will_free) {
        derived::FreeState(&src);
    }
}
template <typename derived_, typename calc_type_>
void GPUVectorPolicyBase<derived_, calc_type_>::ApplySingleQubitMatrix(const qs_data_p_t& src_out, qs_data_p_t* des_p,
                                                                       qbit_t obj_qubit, const qbits_t& ctrls,
                                                                       const std::vector<std::vector<py_qs_data_t>>& m,
                                                                       index_t dim) {
    auto& des = (*des_p);
    if (des == nullptr) {
        des = derived::InitState(dim);
    }
    qs_data_p_t src;
    bool will_free = false;
    if (src_out == nullptr) {
        src = derived::InitState(dim);
        will_free = true;
    } else {
        src = src_out;
    }
    SingleQubitGateMask mask({obj_qubit}, ctrls);
    qs_data_t m00 = m[0][0];
    qs_data_t m01 = m[0][1];
    qs_data_t m10 = m[1][0];
    qs_data_t m11 = m[1][1];
    auto obj_high_mask = mask.obj_high_mask;
    auto obj_low_mask = mask.obj_low_mask;
    auto obj_mask = mask.obj_mask;
    auto ctrl_mask = mask.ctrl_mask;
    thrust::counting_iterator<size_t> l(0);
    if (!mask.ctrl_mask) {
        thrust::for_each(l, l + (dim / 2), [=] __device__(size_t l) {
            auto i = ((l & obj_high_mask) << 1) + (l & obj_low_mask);
            auto j = i + obj_mask;
            auto a = m00 * src[i] + m01 * src[j];
            auto b = m10 * src[i] + m11 * src[j];
            des[i] = a;
            des[j] = b;
        });
    } else {
        thrust::for_each(l, l + (dim / 2), [=] __device__(size_t l) {
            auto i = ((l & obj_high_mask) << 1) + (l & obj_low_mask);
            if ((i & ctrl_mask) == ctrl_mask) {
                auto j = i + obj_mask;
                auto a = m00 * src[i] + m01 * src[j];
                auto b = m10 * src[i] + m11 * src[j];
                des[i] = a;
                des[j] = b;
            }
        });
    }
    if (will_free) {
        derived_::FreeState(&src);
    }
}

template <typename derived_, typename calc_type_>
void GPUVectorPolicyBase<derived_, calc_type_>::ApplyMatrixGate(const qs_data_p_t& src, qs_data_p_t* des_p,
                                                                const qbits_t& objs, const qbits_t& ctrls,
                                                                const std::vector<std::vector<py_qs_data_t>>& m,
                                                                index_t dim) {
    if (objs.size() == 1) {
        derived::ApplySingleQubitMatrix(src, des_p, objs[0], ctrls, m, dim);
    } else if (objs.size() == 2) {
        derived::ApplyTwoQubitsMatrix(src, des_p, objs, ctrls, m, dim);
    } else {
        derived::ApplyNQubitsMatrix(src, des_p, objs, ctrls, m, dim);
    }
}

template struct GPUVectorPolicyBase<CuQuantumVectorPolicyFloat, float>;
template struct GPUVectorPolicyBase<CuQuantumVectorPolicyDouble, double>;
template struct GPUVectorPolicyBase<GPUVectorPolicyFloat, float>;
template struct GPUVectorPolicyBase<GPUVectorPolicyDouble, double>;

}  // namespace mindquantum::sim::vector::detail
